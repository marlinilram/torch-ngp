#include <stdint.h>
#include <hip/hip_runtime.h>


#include <algorithm>
#include <stdexcept>

#include <cstdio>

template <typename T>
__host__ __device__ T div_round_up(T val, T divisor) {
	return (val + divisor - 1) / divisor;
}


template <uint32_t D>
__device__ uint32_t fast_hash(const uint32_t pos_grid[D]) {
	static_assert(D <= 7, "fast_hash can only hash up to 7 dimensions.");

	// While 1 is technically not a good prime for hashing (or a prime at all), it helps memory coherence
	// and is sufficient for our use case of obtaining a uniformly colliding index from high-dimensional
	// coordinates.
	constexpr uint32_t primes[7] = { 1, 19349663, 83492791, 25165843, 6291469, 12582917, 3145739 };

	uint32_t result = 0;
	#pragma unroll
	for (uint32_t i = 0; i < D; ++i) {
		result ^= pos_grid[i] * primes[i];
	}

	return result;
}


template <uint32_t D, uint32_t C>
__device__ uint32_t get_grid_index(const uint32_t ch, const uint32_t hashmap_size, const uint32_t resolution, const uint32_t pos_grid[D]) {
	uint32_t stride = 1;
	uint32_t index = 0;

	#pragma unroll
    for (uint32_t d = 0; d < D && stride <= hashmap_size; d++) {
        //printf("get_grid_index d=%d, pos_grid[d]=%d, stride=%d, reso=%d\n", d, pos_grid[d], stride, resolution);
        index += pos_grid[d] * stride;
        stride *= (resolution + 1);
    }

    if (stride > hashmap_size) {
        //printf("hash because %d > %d\n", stride, hashmap_size);
        index = fast_hash<D>(pos_grid);
        //printf("hashed (%d, %d) = %d to %d in %d\n", pos_grid[0], pos_grid[1], pos_grid[0] + resolution * pos_grid[1], index % hashmap_size, hashmap_size);
    }

	return (index % hashmap_size) * C + ch;
}


template <uint32_t D, uint32_t C>
__global__ void kernel_grid(
    const float * __restrict__ inputs, 
    const float * __restrict__ grid, 
    const int * __restrict__ offsets, 
    float * outputs, 
    uint32_t B, uint32_t L, uint32_t H,
    const bool calc_grad_inputs, 
    float * dy_dx
) {
    const uint32_t b = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (b >= B) return;

    const uint32_t level = blockIdx.y;
    
    // locate
    grid += (uint32_t)offsets[level] * C;
    inputs += b * D;
    outputs += b * L * C + level * C;

    const uint32_t hashmap_size = offsets[level + 1] - offsets[level];
    const float scale = exp2f(level) * H - 1.0f;
    const uint32_t resolution = (uint32_t)ceil(scale) + 1;
    
    // calculate coordinate
    float pos[D];
    uint32_t pos_grid[D];

    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        pos[d] = inputs[d] * scale + 0.5f;
        pos_grid[d] = floorf(pos[d]);
        pos[d] -= (float)pos_grid[d];
    }

    //printf("[b=%d, l=%d] pos=(%f, %f)+(%d, %d)\n", b, level, pos[0], pos[1], pos_grid[0], pos_grid[1]);

    // interpolate
    #pragma unroll
    for (uint32_t idx = 0; idx < (1 << D); idx++) {
        float w = 1;
        uint32_t pos_grid_local[D];

        #pragma unroll
        for (uint32_t d = 0; d < D; d++) {
            if ((idx & (1 << d)) == 0) {
                w *= 1 - pos[d];
                pos_grid_local[d] = pos_grid[d];
            } else {
                w *= pos[d];
                pos_grid_local[d] = pos_grid[d] + 1;
            }
        }

        uint32_t index = get_grid_index<D, C>(0, hashmap_size, resolution, pos_grid_local);

        #pragma unroll
        for (uint32_t ch = 0; ch < C; ch++) {
            outputs[ch] += w * grid[index + ch];
        }

        //printf("[b=%d, l=%d] int %d, idx %d, w %f, val %f\n", b, level, idx, index, w, grid[index]);
    }    

    // prepare dy_dx for calc_grad_inputs
    if (calc_grad_inputs) {

        dy_dx += b * D * L * C + level * D * C; // B L D C

        #pragma unroll
        for (uint32_t gd = 0; gd < D; gd++) {

            #pragma unroll
            for (uint32_t idx = 0; idx < (1 << (D - 1)); idx++) {
                float w = scale;
                uint32_t pos_grid_local[D];

                #pragma unroll
                for (uint32_t nd = 0; nd < D - 1; nd++) {
                    const uint32_t d = nd > gd ? nd + 1 : nd;

                    if ((idx & (1 << nd)) == 0) {
                        w *= 1 - pos[d];
                        pos_grid_local[d] = pos_grid[d];
                    } else {
                        w *= pos[d];
                        pos_grid_local[d] = pos_grid[d] + 1;
                    }
                }

                pos_grid_local[gd] = pos_grid[gd];
                uint32_t index_left = get_grid_index<D, C>(0, hashmap_size, resolution, pos_grid_local);
                pos_grid_local[gd] = pos_grid[gd] + 1;
                uint32_t index_right = get_grid_index<D, C>(0, hashmap_size, resolution, pos_grid_local);

                #pragma unroll
                for (uint32_t ch = 0; ch < C; ch++) {
                    dy_dx[gd * C + ch] += w * (grid[index_right + ch] - grid[index_left + ch]);
                }
            }
        }
    }
}


template <uint32_t D, uint32_t C, uint32_t N_C>
__global__ void kernel_grid_backward(
    const float * __restrict__ grad,
    const float * __restrict__ inputs, 
    const float * __restrict__ grid, 
    const int * __restrict__ offsets, 
    float * grad_grid, 
    uint32_t B, uint32_t L, uint32_t H
) {
    const uint32_t b = (blockIdx.x * blockDim.x + threadIdx.x) * N_C / C;
	if (b >= B) return;

    const uint32_t level = blockIdx.y;
    const uint32_t ch = (blockIdx.x * blockDim.x + threadIdx.x) * N_C - b * C;

    // locate
    grad_grid += offsets[level] * C;
    inputs += b * D;
    grad += b * L * C + level * C + ch;

    const uint32_t hashmap_size = offsets[level + 1] - offsets[level];
    const float scale = exp2f(level) * H - 1.0f;
    const uint32_t resolution = (uint32_t)ceil(scale) + 1;

    // calculate coordinate
    float pos[D];
    uint32_t pos_grid[D];

    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        pos[d] = inputs[d] * scale + 0.5f;
        pos_grid[d] = floorf(pos[d]);
        pos[d] -= (float)pos_grid[d];
    }

    // interpolate
    #pragma unroll
    for (uint32_t idx = 0; idx < (1 << D); idx++) {
        float w = 1;
        uint32_t pos_grid_local[D];

        #pragma unroll
        for (uint32_t d = 0; d < D; d++) {
            if ((idx & (1 << d)) == 0) {
                w *= 1 - pos[d];
                pos_grid_local[d] = pos_grid[d];
            } else {
                w *= pos[d];
                pos_grid_local[d] = pos_grid[d] + 1;
            }
        }

        uint32_t index = get_grid_index<D, C>(ch, hashmap_size, resolution, pos_grid_local);

        #pragma unroll
        for (uint32_t c = 0; c < N_C; c++) {
            atomicAdd(&grad_grid[index + c], w * grad[c]);
        }
    }    
}


template <uint32_t D, uint32_t C>
__global__ void kernel_input_backward(
    const float * __restrict__ grad,
    const float * __restrict__ dy_dx,  
    float * grad_inputs, 
    uint32_t B, uint32_t L
) {
    const uint32_t t = threadIdx.x + blockIdx.x * blockDim.x;
    if (t >= B * D) return;

    const uint32_t b = t / D;
    const uint32_t d = t - b * D;

    grad += b * L * C;
    dy_dx += b * L * D * C;
    
    # pragma unroll
    for (int l = 0; l < L; l++) {
        # pragma unroll
        for (int ch = 0; ch < C; ch++) {
            grad_inputs[t] += grad[l * C + ch] * dy_dx[l * D * C + d * C + ch];
        }
    }
}


template <uint32_t D>
void kernel_grid_wrapper(const float *inputs, const float *embeddings, const int *offsets, float *outputs, const uint32_t B, const uint32_t C, const uint32_t L, const uint32_t H, const bool calc_grad_inputs, float *dy_dx) {
    static constexpr uint32_t N_THREAD = 512;
	const dim3 blocks_hashgrid = { div_round_up(B, N_THREAD), L, 1 };
    switch (C) {
        case 1: kernel_grid<D, 1><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, H, calc_grad_inputs, dy_dx); break;
        case 2: kernel_grid<D, 2><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, H, calc_grad_inputs, dy_dx); break;
        case 4: kernel_grid<D, 4><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, H, calc_grad_inputs, dy_dx); break;
        case 8: kernel_grid<D, 8><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, H, calc_grad_inputs, dy_dx); break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, or 8."};
    }
}

// inputs: [B, D], float, in [0, 1]
// embeddings: [sO, C], float
// offsets: [L + 1], uint32_t
// outputs: [B, L * C], float
// H: base resolution
void hash_encode_forward_cuda(const float *inputs, const float *embeddings, const int *offsets, float *outputs, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const uint32_t H, const bool calc_grad_inputs, float *dy_dx) {
    switch (D) {
        case 2: kernel_grid_wrapper<2>(inputs, embeddings, offsets, outputs, B, C, L, H, calc_grad_inputs, dy_dx); break;
        case 3: kernel_grid_wrapper<3>(inputs, embeddings, offsets, outputs, B, C, L, H, calc_grad_inputs, dy_dx); break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, or 8."};
    }
    
}

template <uint32_t D>
void kernel_grid_backward_wrapper(const float *grad, const float *inputs, const float *embeddings, const int *offsets, float *grad_embeddings, const uint32_t B, const uint32_t C, const uint32_t L, const uint32_t H, const bool calc_grad_inputs, float *dy_dx, float *grad_inputs) {
    static constexpr uint32_t N_THREAD = 256;
	const uint32_t N_C = std::min(2u, C); // n_features_per_thread
	const dim3 blocks_hashgrid = { div_round_up(B * C / N_C, N_THREAD), L, 1 };
    switch (C) {
        case 1: 
            kernel_grid_backward<D, 1, 1><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, H); 
            if (calc_grad_inputs) kernel_input_backward<D, 1><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 2: 
            kernel_grid_backward<D, 2, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, H);
            if (calc_grad_inputs) kernel_input_backward<D, 2><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 4: 
            kernel_grid_backward<D, 4, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, H);
            if (calc_grad_inputs) kernel_input_backward<D, 4><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 8: 
            kernel_grid_backward<D, 8, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, H);
            if (calc_grad_inputs) kernel_input_backward<D, 8><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, or 8."};
    }
}


// grad: [B, L * C], float
// inputs: [B, D], float, in [0, 1]
// embeddings: [sO, C], float
// offsets: [L + 1], uint32_t
// grad_embeddings: [sO, C]
// H: base resolution
void hash_encode_backward_cuda(const float *grad, const float *inputs, const float *embeddings, const int *offsets, float *grad_embeddings, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const uint32_t H, const bool calc_grad_inputs, float *dy_dx, float *grad_inputs) {
    switch (D) {
        case 2: kernel_grid_backward_wrapper<2>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, H, calc_grad_inputs, dy_dx, grad_inputs); break;
        case 3: kernel_grid_backward_wrapper<3>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, H, calc_grad_inputs, dy_dx, grad_inputs); break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, or 8."};
    }
}
